#include "hip/hip_runtime.h"
#include "../compatibility.cuh"
#include "../cuda_utils.cuh"
#include <float.h>
#include <limits.h>
#include <stdint.h>

#define MAX_DIMS 10

// Helper kernel to initialize output arrays
template <typename T>
__global__ void fill_kernel(T *data, T value, size_t size) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    data[idx] = value;
  }
}

#define SUM_OP(TYPENAME, FN_NAME)                                              \
  extern "C" __global__ void cuda_##FN_NAME##_kernel(                          \
      const size_t num_els, const size_t num_dims, const size_t num_sum_dims,  \
      const size_t *metadata, const TYPENAME *inp, TYPENAME *out) {            \
    const size_t *dims = metadata;                                             \
    const size_t *strides = metadata + num_dims;                               \
    const size_t *sum_dims_l = metadata + 2 * num_dims;                        \
    const size_t *sum_dims_s = metadata + 2 * num_dims + num_sum_dims;         \
    const size_t offset = *(metadata + 2 * num_dims + 2 * num_sum_dims);       \
                                                                               \
    if (is_contiguous(num_dims, dims, strides)) {                              \
      for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;             \
           i < num_els; i += blockDim.x * gridDim.x) {                         \
        size_t dst_index = i;                                                  \
        for (unsigned int nd = 0; nd < num_sum_dims; ++nd) {                   \
          size_t stride = sum_dims_s[nd];                                      \
          size_t pre = dst_index / stride;                                     \
          size_t post = dst_index % stride;                                    \
          dst_index = (pre / sum_dims_l[nd]) * stride + post;                  \
        }                                                                      \
        atomicAdd(out + dst_index, inp[offset + i]);                           \
      }                                                                        \
    } else {                                                                   \
      for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;             \
           i < num_els; i += blockDim.x * gridDim.x) {                         \
        unsigned strided_i =                                                   \
            offset + get_strided_index(i, num_dims, dims, strides);            \
        size_t dst_index = i;                                                  \
        for (unsigned int nd = 0; nd < num_sum_dims; ++nd) {                   \
          size_t stride = sum_dims_s[nd];                                      \
          size_t pre = dst_index / stride;                                     \
          size_t post = dst_index % stride;                                    \
          dst_index = (pre / sum_dims_l[nd]) * stride + post;                  \
        }                                                                      \
        atomicAdd(out + dst_index, inp[strided_i]);                            \
      }                                                                        \
    }                                                                          \
  }                                                                            \
  extern "C" void cuda_##FN_NAME(size_t num_els, size_t num_dims,              \
                                 size_t num_red_dims, const size_t *metadata,  \
                                 const TYPENAME *inp, TYPENAME *out) {         \
    dim3 block_dim(256);                                                       \
    dim3 grid_dim((num_els + block_dim.x - 1) / block_dim.x);                  \
    cuda_##FN_NAME##_kernel<<<grid_dim, block_dim>>>(                          \
        num_els, num_dims, num_red_dims, metadata, inp, out);                  \
  }

#define SUM_TO_SHAPE_OP(TYPENAME, FN_NAME)                                     \
  extern "C" __global__ void cuda_##FN_NAME##_kernel(                          \
      const size_t num_els, const size_t num_dims, const size_t *metadata,     \
      const TYPENAME *inp, TYPENAME *out) {                                    \
    const size_t *input_dims = metadata;                                       \
    const size_t *input_strides = metadata + num_dims;                         \
    const size_t *output_dims = metadata + 2 * num_dims;                       \
    const size_t offset = *(metadata + 3 * num_dims);                          \
                                                                               \
    if (num_dims > MAX_DIMS)                                                   \
      return;                                                                  \
                                                                               \
    size_t reduction_factors[MAX_DIMS];                                        \
    for (size_t d = 0; d < num_dims; d++) {                                    \
      reduction_factors[d] = input_dims[d] / output_dims[d];                   \
    }                                                                          \
                                                                               \
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < num_els;  \
         i += blockDim.x * gridDim.x) {                                        \
                                                                               \
      size_t coords[MAX_DIMS];                                                 \
      size_t tmp_i = i;                                                        \
      for (int d = num_dims - 1; d >= 0; --d) {                                \
        coords[d] = tmp_i % input_dims[d];                                     \
        tmp_i /= input_dims[d];                                                \
      }                                                                        \
                                                                               \
      size_t dst_idx = 0;                                                      \
      for (size_t d = 0; d < num_dims; d++) {                                  \
        size_t out_coord = coords[d] / reduction_factors[d];                   \
        dst_idx = dst_idx * output_dims[d] + out_coord;                        \
      }                                                                        \
                                                                               \
      size_t src_idx = offset;                                                 \
      for (size_t d = 0; d < num_dims; d++) {                                  \
        src_idx += coords[d] * input_strides[d];                               \
      }                                                                        \
                                                                               \
      atomicAdd(out + dst_idx, inp[src_idx]);                                  \
    }                                                                          \
  }                                                                            \
  extern "C" void cuda_##FN_NAME(size_t num_els, size_t num_dims,              \
                                 const size_t *metadata, const TYPENAME *inp,  \
                                 TYPENAME *out) {                              \
    dim3 block_dim(256);                                                       \
    dim3 grid_dim((num_els + block_dim.x - 1) / block_dim.x);                  \
    cuda_##FN_NAME##_kernel<<<grid_dim, block_dim>>>(num_els, num_dims,        \
                                                     metadata, inp, out);      \
  }

#define MEAN_OP(TYPENAME, FN_NAME)                                             \
  extern "C" __global__ void cuda_##FN_NAME##_kernel(                          \
      const size_t num_els, const size_t num_dims, const size_t num_mean_dims, \
      const size_t *metadata, const TYPENAME *inp, TYPENAME *out) {            \
    const size_t *dims = metadata;                                             \
    const size_t *strides = metadata + num_dims;                               \
    const size_t *mean_dims_l = metadata + 2 * num_dims;                       \
    const size_t *mean_dims_s = metadata + 2 * num_dims + num_mean_dims;       \
    const size_t offset = *(metadata + 2 * num_dims + 2 * num_mean_dims);      \
                                                                               \
    /* Calculate reduction factor */                                           \
    size_t reduction_factor = 1;                                               \
    for (size_t i = 0; i < num_mean_dims; i++) {                               \
      reduction_factor *= mean_dims_l[i];                                      \
    }                                                                          \
    TYPENAME factor = static_cast<TYPENAME>(reduction_factor);                 \
                                                                               \
    if (is_contiguous(num_dims, dims, strides)) {                              \
      for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;             \
           i < num_els; i += blockDim.x * gridDim.x) {                         \
        size_t dst_index = i;                                                  \
        for (unsigned int nd = 0; nd < num_mean_dims; ++nd) {                  \
          size_t stride = mean_dims_s[nd];                                     \
          size_t pre = dst_index / stride;                                     \
          size_t post = dst_index % stride;                                    \
          dst_index = (pre / mean_dims_l[nd]) * stride + post;                 \
        }                                                                      \
        atomicAdd(out + dst_index, inp[offset + i] / factor);                  \
      }                                                                        \
    } else {                                                                   \
      for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;             \
           i < num_els; i += blockDim.x * gridDim.x) {                         \
        unsigned strided_i =                                                   \
            offset + get_strided_index(i, num_dims, dims, strides);            \
        size_t dst_index = i;                                                  \
        for (unsigned int nd = 0; nd < num_mean_dims; ++nd) {                  \
          size_t stride = mean_dims_s[nd];                                     \
          size_t pre = dst_index / stride;                                     \
          size_t post = dst_index % stride;                                    \
          dst_index = (pre / mean_dims_l[nd]) * stride + post;                 \
        }                                                                      \
        atomicAdd(out + dst_index, inp[strided_i] / factor);                   \
      }                                                                        \
    }                                                                          \
  }                                                                            \
  extern "C" void cuda_##FN_NAME(size_t num_els, size_t num_dims,              \
                                 size_t num_red_dims, const size_t *metadata,  \
                                 const TYPENAME *inp, TYPENAME *out) {         \
    dim3 block_dim(256);                                                       \
    dim3 grid_dim((num_els + block_dim.x - 1) / block_dim.x);                  \
    cuda_##FN_NAME##_kernel<<<grid_dim, block_dim>>>(                          \
        num_els, num_dims, num_red_dims, metadata, inp, out);                  \
  }

#define FOLD_OP(TYPENAME, FN_NAME)                                             \
  extern "C" __global__ void cuda_##FN_NAME##_kernel(                          \
      const size_t num_els, const size_t num_dims, const size_t *metadata,     \
      const TYPENAME *inp, TYPENAME *out) {                                    \
    const size_t *input_dims = metadata;                                       \
    const size_t *input_strides = metadata + num_dims;                         \
    const size_t fold_dim = *(metadata + 2 * num_dims);                        \
    const size_t window_dim = *(metadata + 2 * num_dims + 1);                  \
    const size_t fold_size = *(metadata + 2 * num_dims + 2);                   \
    const size_t step = *(metadata + 2 * num_dims + 3);                        \
    const size_t window_size = *(metadata + 2 * num_dims + 4);                 \
    const size_t offset = *(metadata + 2 * num_dims + 5);                      \
                                                                               \
    if (num_dims > MAX_DIMS)                                                   \
      return;                                                                  \
                                                                               \
    /* Process elements in parallel */                                         \
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < num_els;  \
         i += blockDim.x * gridDim.x) {                                        \
                                                                               \
      /* Calculate coordinates in input tensor */                              \
      size_t coords[MAX_DIMS];                                                 \
      size_t tmp_i = i;                                                        \
      for (int d = num_dims - 1; d >= 0; --d) {                                \
        coords[d] = tmp_i % input_dims[d];                                     \
        tmp_i /= input_dims[d];                                                \
      }                                                                        \
                                                                               \
      /* Calculate source index using input strides */                         \
      size_t src_idx = offset;                                                 \
      for (size_t d = 0; d < num_dims; d++) {                                  \
        src_idx += coords[d] * input_strides[d];                               \
      }                                                                        \
                                                                               \
      /* Extract window index and position in window */                        \
      const size_t window_idx = coords[fold_dim];                              \
      const size_t pos_in_window = coords[window_dim];                         \
                                                                               \
      /* Calculate position in the original folded dimension */                \
      const size_t orig_pos = window_idx * step + pos_in_window;               \
                                                                               \
      /* Skip if outside the bounds of the folded dimension */                 \
      if (orig_pos >= fold_size) {                                             \
        continue;                                                              \
      }                                                                        \
                                                                               \
      /* Calculate destination index in output */                              \
      size_t dst_idx = 0;                                                      \
      size_t dst_dim_idx = 0;                                                  \
                                                                               \
      for (size_t d = 0; d < num_dims; d++) {                                  \
        if (d == window_dim) {                                                 \
          continue; /* Skip window dimension */                                \
        } else if (d == fold_dim) {                                            \
          dst_idx = dst_idx * fold_size + orig_pos;                            \
        } else {                                                               \
          dst_idx = dst_idx * input_dims[d] + coords[d];                       \
        }                                                                      \
      }                                                                        \
                                                                               \
      /* Add value to output */                                                \
      atomicAdd(out + dst_idx, inp[src_idx]);                                  \
    }                                                                          \
  }                                                                            \
                                                                               \
  extern "C" void cuda_##FN_NAME(size_t num_els, size_t num_dims,              \
                                 const size_t *metadata, const TYPENAME *inp,  \
                                 TYPENAME *out) {                              \
    dim3 block_dim(256);                                                       \
    dim3 grid_dim((num_els + block_dim.x - 1) / block_dim.x);                  \
    cuda_##FN_NAME##_kernel<<<grid_dim, block_dim>>>(num_els, num_dims,        \
                                                     metadata, inp, out);      \
  }

#define MAX_OP(TYPENAME, FN_NAME, MIN_VALUE)                                   \
  extern "C" __global__ void cuda_##FN_NAME##_kernel(                          \
      const size_t num_els, const size_t num_dims, const size_t num_max_dims,  \
      const size_t *metadata, const TYPENAME *inp, TYPENAME *out) {            \
    const size_t *dims = metadata;                                             \
    const size_t *strides = metadata + num_dims;                               \
    const size_t *max_dims_l = metadata + 2 * num_dims;                        \
    const size_t *max_dims_s = metadata + 2 * num_dims + num_max_dims;         \
    const size_t offset = *(metadata + 2 * num_dims + 2 * num_max_dims);       \
                                                                               \
    /* Calculate output size */                                                \
    size_t out_size = num_els;                                                 \
    for (size_t i = 0; i < num_max_dims; i++) {                                \
      out_size /= max_dims_l[i];                                               \
    }                                                                          \
                                                                               \
    /* Initialize output with minimum possible values */                       \
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < out_size; \
         i += blockDim.x * gridDim.x) {                                        \
      out[i] = MIN_VALUE;                                                      \
    }                                                                          \
    __syncthreads();                                                           \
                                                                               \
    bool is_contiguous = true;                                                 \
    size_t acc = 1;                                                            \
    for (int d = num_dims - 1; d >= 0; d--) {                                  \
      if (strides[d] != acc) {                                                 \
        is_contiguous = false;                                                 \
        break;                                                                 \
      }                                                                        \
      acc *= dims[d];                                                          \
    }                                                                          \
                                                                               \
    /* Process elements */                                                     \
    if (is_contiguous) {                                                       \
      for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;             \
           i < num_els; i += blockDim.x * gridDim.x) {                         \
        size_t src_idx = i;                                                    \
        size_t src_value_idx = (src_idx + offset) % num_els;                   \
                                                                               \
        /* Calculate destination index */                                      \
        size_t dst_idx = i;                                                    \
        for (unsigned int nd = 0; nd < num_max_dims; ++nd) {                   \
          size_t stride = max_dims_s[nd];                                      \
          size_t pre = dst_idx / stride;                                       \
          size_t post = dst_idx % stride;                                      \
          dst_idx = (pre / max_dims_l[nd]) * stride + post;                    \
        }                                                                      \
                                                                               \
        /* Update max value atomically */                                      \
        atomicMax(out + dst_idx, inp[src_value_idx]);                          \
      }                                                                        \
    } else {                                                                   \
      for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;             \
           i < num_els; i += blockDim.x * gridDim.x) {                         \
        size_t src_idx = get_strided_index(i, num_dims, dims, strides);        \
        size_t src_value_idx = (src_idx + offset) % num_els;                   \
                                                                               \
        /* Calculate destination index */                                      \
        size_t dst_idx = i;                                                    \
        for (unsigned int nd = 0; nd < num_max_dims; ++nd) {                   \
          size_t stride = max_dims_s[nd];                                      \
          size_t pre = dst_idx / stride;                                       \
          size_t post = dst_idx % stride;                                      \
          dst_idx = (pre / max_dims_l[nd]) * stride + post;                    \
        }                                                                      \
                                                                               \
        /* Update max value atomically */                                      \
        atomicMax(out + dst_idx, inp[src_value_idx]);                          \
      }                                                                        \
    }                                                                          \
  }                                                                            \
  extern "C" void cuda_##FN_NAME(size_t num_els, size_t num_dims,              \
                                 size_t num_red_dims, const size_t *metadata,  \
                                 const TYPENAME *inp, TYPENAME *out) {         \
    /* Calculate output size */                                                \
    size_t out_size = num_els;                                                 \
    const size_t *max_dims_l = metadata + 2 * num_dims;                        \
    for (size_t i = 0; i < num_red_dims; i++) {                                \
      out_size /= max_dims_l[i];                                               \
    }                                                                          \
                                                                               \
    dim3 block_dim(256);                                                       \
    unsigned int grid_size = (num_els + block_dim.x - 1) / block_dim.x;        \
    grid_size = (grid_size > 65535u) ? 65535u : grid_size;                     \
    dim3 grid_dim(grid_size);                                                  \
    cuda_##FN_NAME##_kernel<<<grid_dim, block_dim>>>(                          \
        num_els, num_dims, num_red_dims, metadata, inp, out);                  \
  }

#define MIN_OP(TYPENAME, FN_NAME, MAX_VALUE)                                   \
  extern "C" __global__ void cuda_##FN_NAME##_kernel(                          \
      const size_t num_els, const size_t num_dims, const size_t num_min_dims,  \
      const size_t *metadata, const TYPENAME *inp, TYPENAME *out) {            \
    const size_t *dims = metadata;                                             \
    const size_t *strides = metadata + num_dims;                               \
    const size_t *min_dims_l = metadata + 2 * num_dims;                        \
    const size_t *min_dims_s = metadata + 2 * num_dims + num_min_dims;         \
    const size_t offset = *(metadata + 2 * num_dims + 2 * num_min_dims);       \
                                                                               \
    /* Calculate output size */                                                \
    size_t out_size = num_els;                                                 \
    for (size_t i = 0; i < num_min_dims; i++) {                                \
      out_size /= min_dims_l[i];                                               \
    }                                                                          \
                                                                               \
    /* Initialize output with maximum possible values */                       \
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < out_size; \
         i += blockDim.x * gridDim.x) {                                        \
      out[i] = MAX_VALUE;                                                      \
    }                                                                          \
    __syncthreads();                                                           \
                                                                               \
    bool is_contiguous = true;                                                 \
    size_t acc = 1;                                                            \
    for (int d = num_dims - 1; d >= 0; d--) {                                  \
      if (strides[d] != acc) {                                                 \
        is_contiguous = false;                                                 \
        break;                                                                 \
      }                                                                        \
      acc *= dims[d];                                                          \
    }                                                                          \
                                                                               \
    /* Process elements */                                                     \
    if (is_contiguous) {                                                       \
      for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;             \
           i < num_els; i += blockDim.x * gridDim.x) {                         \
        size_t src_idx = i;                                                    \
        size_t src_value_idx = (src_idx + offset) % num_els;                   \
                                                                               \
        /* Calculate destination index */                                      \
        size_t dst_idx = i;                                                    \
        for (unsigned int nd = 0; nd < num_min_dims; ++nd) {                   \
          size_t stride = min_dims_s[nd];                                      \
          size_t pre = dst_idx / stride;                                       \
          size_t post = dst_idx % stride;                                      \
          dst_idx = (pre / min_dims_l[nd]) * stride + post;                    \
        }                                                                      \
                                                                               \
        /* Update min value atomically */                                      \
        atomicMin(out + dst_idx, inp[src_value_idx]);                          \
      }                                                                        \
    } else {                                                                   \
      for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;             \
           i < num_els; i += blockDim.x * gridDim.x) {                         \
        size_t src_idx = get_strided_index(i, num_dims, dims, strides);        \
        size_t src_value_idx = (src_idx + offset) % num_els;                   \
                                                                               \
        /* Calculate destination index */                                      \
        size_t dst_idx = i;                                                    \
        for (unsigned int nd = 0; nd < num_min_dims; ++nd) {                   \
          size_t stride = min_dims_s[nd];                                      \
          size_t pre = dst_idx / stride;                                       \
          size_t post = dst_idx % stride;                                      \
          dst_idx = (pre / min_dims_l[nd]) * stride + post;                    \
        }                                                                      \
                                                                               \
        /* Update min value atomically */                                      \
        atomicMin(out + dst_idx, inp[src_value_idx]);                          \
      }                                                                        \
    }                                                                          \
  }                                                                            \
  extern "C" void cuda_##FN_NAME(size_t num_els, size_t num_dims,              \
                                 size_t num_red_dims, const size_t *metadata,  \
                                 const TYPENAME *inp, TYPENAME *out) {         \
    /* Calculate output size */                                                \
    size_t out_size = num_els;                                                 \
    const size_t *min_dims_l = metadata + 2 * num_dims;                        \
    for (size_t i = 0; i < num_red_dims; i++) {                                \
      out_size /= min_dims_l[i];                                               \
    }                                                                          \
                                                                               \
    dim3 block_dim(256);                                                       \
    unsigned int grid_size = (num_els + block_dim.x - 1) / block_dim.x;        \
    grid_size = (grid_size > 65535u) ? 65535u : grid_size;                     \
    dim3 grid_dim(grid_size);                                                  \
    cuda_##FN_NAME##_kernel<<<grid_dim, block_dim>>>(                          \
        num_els, num_dims, num_red_dims, metadata, inp, out);                  \
  }

SUM_OP(float, sum_f32);
SUM_OP(double, sum_f64);
SUM_OP(uint8_t, sum_u8);
SUM_OP(uint16_t, sum_u16);
SUM_OP(uint32_t, sum_u32);
SUM_OP(uint64_t, sum_u64);
SUM_OP(int8_t, sum_i8);
SUM_OP(int16_t, sum_i16);
SUM_OP(int32_t, sum_i32);
SUM_OP(int64_t, sum_i64);

SUM_TO_SHAPE_OP(float, sum_to_shape_f32);
SUM_TO_SHAPE_OP(double, sum_to_shape_f64);
SUM_TO_SHAPE_OP(uint8_t, sum_to_shape_u8);
SUM_TO_SHAPE_OP(uint16_t, sum_to_shape_u16);
SUM_TO_SHAPE_OP(uint32_t, sum_to_shape_u32);
SUM_TO_SHAPE_OP(uint64_t, sum_to_shape_u64);
SUM_TO_SHAPE_OP(int8_t, sum_to_shape_i8);
SUM_TO_SHAPE_OP(int16_t, sum_to_shape_i16);
SUM_TO_SHAPE_OP(int32_t, sum_to_shape_i32);
SUM_TO_SHAPE_OP(int64_t, sum_to_shape_i64);

MEAN_OP(float, mean_f32);
MEAN_OP(double, mean_f64);

FOLD_OP(float, fold_f32);
FOLD_OP(double, fold_f64);
FOLD_OP(uint8_t, fold_u8);
FOLD_OP(uint16_t, fold_u16);
FOLD_OP(uint32_t, fold_u32);
FOLD_OP(uint64_t, fold_u64);
FOLD_OP(int8_t, fold_i8);
FOLD_OP(int16_t, fold_i16);
FOLD_OP(int32_t, fold_i32);
FOLD_OP(int64_t, fold_i64);

MAX_OP(float, max_f32, -FLT_MAX);
MAX_OP(double, max_f64, -DBL_MAX);
MAX_OP(uint8_t, max_u8, 0);
MAX_OP(uint16_t, max_u16, 0);
MAX_OP(uint32_t, max_u32, 0);
MAX_OP(uint64_t, max_u64, 0);
MAX_OP(int8_t, max_i8, INT8_MIN);
MAX_OP(int16_t, max_i16, INT16_MIN);
MAX_OP(int32_t, max_i32, INT32_MIN);
MAX_OP(int64_t, max_i64, INT64_MIN);

MIN_OP(float, min_f32, FLT_MAX);
MIN_OP(double, min_f64, DBL_MAX);
MIN_OP(uint8_t, min_u8, UINT8_MAX);
MIN_OP(uint16_t, min_u16, UINT16_MAX);
MIN_OP(uint32_t, min_u32, UINT32_MAX);
MIN_OP(uint64_t, min_u64, UINT64_MAX);
MIN_OP(int8_t, min_i8, INT8_MAX);
MIN_OP(int16_t, min_i16, INT16_MAX);
MIN_OP(int32_t, min_i32, INT32_MAX);
MIN_OP(int64_t, min_i64, INT64_MAX);

// __half
SUM_OP(__half, sum_f16);
SUM_TO_SHAPE_OP(__half, sum_to_shape_f16);
MEAN_OP(__half, mean_f16);
FOLD_OP(__half, fold_f16);
MAX_OP(__half, max_f16, __float2half(-FLT_MAX));
MIN_OP(__half, min_f16, __float2half(FLT_MAX));

// __hip_bfloat16
SUM_OP(__hip_bfloat16, sum_bf16);
SUM_TO_SHAPE_OP(__hip_bfloat16, sum_to_shape_bf16);
MEAN_OP(__hip_bfloat16, mean_bf16);
FOLD_OP(__hip_bfloat16, fold_bf16);
MAX_OP(__hip_bfloat16, max_bf16, __float2bfloat16(-FLT_MAX));
MIN_OP(__hip_bfloat16, min_bf16, __float2bfloat16(FLT_MAX));
