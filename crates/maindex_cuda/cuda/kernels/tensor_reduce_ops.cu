#include "hip/hip_runtime.h"
#include "tensor_reduce_ops.cuh"
#define BLOCK_SIZE 256

__global__ void tensor_mean_kernel(float *output, const float *input,
                                   size_t size) {
  __shared__ float shared_sum[BLOCK_SIZE];

  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  shared_sum[tid] = 0.0f;
  if (idx < size) {
    shared_sum[tid] = input[idx];
  }
  __syncthreads();

  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      shared_sum[tid] += shared_sum[tid + s];
    }
    __syncthreads();
  }

  if (tid == 0) {
    atomicAdd(output, shared_sum[0] / size);
  }
}

__global__ void tensor_sum_kernel(float *output, const float *input,
                                  size_t size) {
  __shared__ float shared_sum[BLOCK_SIZE];

  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  shared_sum[tid] = 0.0f;
  if (idx < size) {
    shared_sum[tid] = input[idx];
  }
  __syncthreads();

  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      shared_sum[tid] += shared_sum[tid + s];
    }
    __syncthreads();
  }

  if (tid == 0) {
    atomicAdd(output, shared_sum[0]);
  }
}

extern "C" {
void tensor_mean(float *output, const float *input, size_t size,
                 hipStream_t stream) {
  float zero = 0.0f;
  hipMemcpy(output, &zero, sizeof(float), hipMemcpyHostToDevice);

  int block_size = BLOCK_SIZE;
  int num_blocks = (size + block_size - 1) / block_size;
  tensor_mean_kernel<<<num_blocks, block_size, 0, stream>>>(output, input,
                                                            size);
}
void tensor_sum(float *output, const float *input, size_t size,
                hipStream_t stream) {
  float zero = 0.0f;
  hipMemcpy(output, &zero, sizeof(float), hipMemcpyHostToDevice);

  int block_size = BLOCK_SIZE;
  int num_blocks = (size + block_size - 1) / block_size;
  tensor_sum_kernel<<<num_blocks, block_size, 0, stream>>>(output, input, size);
}
}
